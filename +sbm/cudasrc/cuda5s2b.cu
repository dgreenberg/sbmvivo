#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand/hiprand_kernel.h> //need to include this before cuda5s.h so we don't redefine device prop. struct
#include "cuda5s2b.h"
#include <sys/time.h>
#include <cfloat>
#include <hipcub/hipcub.hpp>

#define RTYPE hiprandStateXORWOW
#define SHAREDMEMRESAMPLE true
#define GUARDFAC 1.0204
#define R_INIT_SD 4.0
#define GUARDMODE true
#define TIMECALC false
//#define logfunc __logf
#define logfunc logf
//#define expfunc __expf
#define expfunc expf

#ifndef INT_MIN
#define INT_MIN -32767
#endif

void cufreeifnotnull(void *&);
timespec tdiff(timespec start, timespec end);
void process_weights(gpu5s_problem *g, int t, int offset);
__global__ void resample_shared(const int * __restrict__ O, int np, int prevoffset, int offset,
		float4 * states_all, float4 * cbr_all, float *log_w, float *log_w_corrected, int *ancestor);
__global__ void advancepf(float fcorr, int nsteps, float stepsize, int n_newton_iterations,
		float4 * states_all, float4 * cbr_all, float * __restrict__ gp,
		bool * __restrict__ ns,
		float * __restrict__ log_wall, float * __restrict__ log_w_correctedall,
		const float * __restrict__ q_spike, RTYPE *rngstates, int np, bool resamplenow, float equal_log_w);

bool resamplenow;
float h_neff;

__device__ float max_log_w, sum_w, max_log_w_corrected, sum_w_corrected, sum_wsq, log_total_sum_weights, log_total_sum_weights_c, d_neff;
__device__ int mlw_total, mlw_c_total;
__device__ unsigned int retirementCount; //for use with threadfence reduction
__constant__ float FBGp1, basevar, gain, p_spike, c0, maxex, kd_ex, sigr_sqrtdt, S;
__constant__ float kon0, kon1, kon2, kon3, koff0, koff1, koff2, koff3;
__constant__ float kon_B0, koff_B0, kon_B1, koff_B1, Btot0, Btot1;
__constant__ float db1, db2, db3, db4;

/* #define CUCALL(val)    cucheck( (val), #val, __FILE__, __LINE__ )
void cucheck(hipError_t  e, char const *const func, const char *const file, int const line) {
	if (e != hipSuccess) {
		std::cerr << "CUDA error at " << file << ":" << line << " code=" << static_cast<unsigned int>(e) << " \"" << func << "\" \n";
		exit(EXIT_FAILURE);
	}
} */

//FIXME decouple reset / list functions from cuda5s code, compile seperately
int cudaDeviceList_wrapper(hipDeviceProp_t *pDeviceList, int MaxDevices) {
	//pDeviceList is a pointer into a pre-initialized array of hipDeviceProp_t structs of size MaxDevices
	//returns number of available devices, or a negative error code
	//for errors during calls to hipGetDeviceProperties, the negative error code is decremented by 1000
	int count = 0;
	hipError_t DeviceCountResult = hipGetDeviceCount ( &count );

	if (DeviceCountResult != hipSuccess) {
		return -((int) DeviceCountResult);
	}

	int n = MIN(count, MaxDevices); //number of devices to query
	for (int j = 0; j < n; j++) {

		hipError_t DevicePropertiesResult = hipGetDeviceProperties (pDeviceList + j, j);
		if (DevicePropertiesResult != hipSuccess) {
			return -1000 - ((int) DevicePropertiesResult);
		}
	}
	return count;
}

void cudaDeviceReset_wrapper() {
	hipDeviceReset();
}

void cudaMemcpy_d2h_wrapper(void *h, void *d, int nbytes) {
	hipMemcpy(h, d, nbytes, hipMemcpyDeviceToHost); //fixme: check for success here
}

void cudaMemcpy_h2d_wrapper(void *d, void *h, int nbytes) {
	hipMemcpy(d, h, nbytes, hipMemcpyHostToDevice); //fixme: check for success here
}

__global__ void fillarray_kernel(float *x, float v, int np) {
	int ii = threadIdx.x + blockIdx.x * BLOCKSIZE;
	while (ii < np) {
		x[ii] = v;
		ii += BLOCKSIZE * gridDim.x; //grid strides
	}
}

void fillarray(float *x, float v, int n) {
	int nblocks = (n % NV) ? n / NV + 1: n / NV;
	fillarray_kernel<<<nblocks, NT>>>(x, v, n);
}

__global__ void initancestors_noresample(int *ancestor, int np) {
	int ii = threadIdx.x + blockIdx.x * BLOCKSIZE;
	while (ii < np) {
		ancestor[ii] = ii; //note that the next time step is the same as K time steps back. it's ok to overwrite this since we've already copied out the relevant values as a_gs
		ii += BLOCKSIZE * gridDim.x;
	}
}

__global__ void W2O(float u, int np, const float * __restrict__ W, int * __restrict__ O) {
	//fixme should combine this kernel with a max-int-scan
	int ii = threadIdx.x + blockIdx.x * BLOCKSIZE;
	float Wfac = ((float) np) / W[np - 1];
	while (ii < np) {
		O[ii] = MIN(np, (int) (W[ii] * Wfac + u));
		ii += BLOCKSIZE * gridDim.x;
	}
}

__global__ void subtract_and_exponentiate_correctedonly(const float* __restrict__ log_w_corrected, float *w_corrected, int np) {
	//fixme should combine this kernel with an add-float-scan
	int ii = threadIdx.x + blockIdx.x * BLOCKSIZE;
	while (ii < np) {
		w_corrected[ii] = expfunc(log_w_corrected[ii] - max_log_w_corrected); //at present, w_corrected is used here only to store linear domain values for a float-add-reduction
		ii += BLOCKSIZE * gridDim.x;
	}
}

float gpu5s_marglik(gpu5s_problem *g) {
	float *ll = (float *) malloc(sizeof(float) * g->T);
	hipMemcpy(ll, g->d.log_sum_raw_w, g->T * sizeof(float), hipMemcpyDeviceToHost); //fixme check for success
	double s = 0.0;
	for (int ii = 0; ii < g->T; ii++) //fixme do this on the gpu
		s += (double) ll[ii];
	free(ll);
	return (float) s;
}

void FreeCubTemporaryArrays(gpu5s_problem *g) {
	cufreeifnotnull(g->d.d_temp_storage_float_max_reduction);
	cufreeifnotnull(g->d.d_temp_storage_float_add_reduction);
	cufreeifnotnull(g->d.d_temp_storage_float_add_scan);
	cufreeifnotnull(g->d.d_temp_storage_uint_max_scan);
}

int InitializeCubTemporaryArrays(gpu5s_problem *g) { //initialize temporary arrays for cub scans and reductions

	//dummy variables for the calls to determine temporary storage requirements
	float *pd_mlw_corrected = NULL;
	float *pd_sw_corrected = NULL;

	//make calls to reduction and scan functions with temp storage pointers set to NULL, to determine storage requirements.
	//no actual scan/reduction work is done by these calls
	//FIXME these temporary arrays should be allocated along with other device memory, not here in the main GPU-PF function
	if (	hipcub::DeviceReduce::Max(NULL, 		 g->d.temp_storage_bytes_float_max_reduction, g->d.log_w_corrected, pd_mlw_corrected, g->options.nparticles) != hipSuccess ||
			hipcub::DeviceReduce::Sum(NULL, 		 g->d.temp_storage_bytes_float_add_reduction, g->d.w_corrected,     pd_sw_corrected,  g->options.nparticles) != hipSuccess ||
			hipcub::DeviceScan::InclusiveSum(NULL,  g->d.temp_storage_bytes_float_add_scan,      g->d.w, 				g->d.W, 	      g->options.nparticles) != hipSuccess ||
			hipcub::DeviceScan::InclusiveScan(NULL, g->d.temp_storage_bytes_uint_max_scan,       g->d.O, 				g->d.Oi, hipcub::Max(), g->options.nparticles)	!= hipSuccess) {
		std::cerr << "failed to determine temporary array requirements for cub\n";
		return -1;
	}

	//allocate temporary arrays of the needed sizes
	if (	hipMalloc((void **) &g->d.d_temp_storage_float_max_reduction, g->d.temp_storage_bytes_float_max_reduction) != hipSuccess ||
			hipMalloc((void **) &g->d.d_temp_storage_float_add_reduction, g->d.temp_storage_bytes_float_add_reduction) != hipSuccess ||
			hipMalloc((void **) &g->d.d_temp_storage_float_add_scan, 	   g->d.temp_storage_bytes_float_add_scan)  	!= hipSuccess ||
			hipMalloc((void **) &g->d.d_temp_storage_uint_max_scan, 	   g->d.temp_storage_bytes_uint_max_scan) 	    != hipSuccess) {
		std::cerr << "failed to initialize one or more arrays requirements for cub\n";
		FreeCubTemporaryArrays(g);
		return -2;
	}
	return 1;
}

int runpfmainloop(gpu5s_problem *g){
	timespec ts, ts2, td, tsum, t0, t1, trand;
	if (TIMECALC)
		clock_gettime(CLOCK_REALTIME, &trand);
	int np = g->options.nparticles;
	int nresamples = 0;
	//get pointers to reduction outputs, which are global device memory variables
	float *pd_mlw_corrected, *pd_sw_corrected;
	hipGetSymbolAddress((void **) &pd_mlw_corrected, max_log_w_corrected);
	hipGetSymbolAddress((void **) &pd_sw_corrected, sum_w_corrected);

	//initialize reduction values
	float z = 0.0; float o = 1.0;
	hipMemcpyToSymbol(HIP_SYMBOL(max_log_w_corrected), &z,  sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sum_w_corrected),     &o,  sizeof(float));
	unsigned int uz = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(retirementCount),     &uz, sizeof(unsigned int));
	float foffset = g->params.fdc;

	int offset_Kbuffers; //offset into array that stores nparticles values buffered K times
	int offset_ancestors; //offset into array that stores ancestors, maxK + 1 columns (a power of 2)
	int offset_ns;  //offset into spiking, K * nsteps columns

	int prevoffset, offset = 0; //offset for the previous and current time steps into buffers with 2 * nparticles capacity
	float stepsize    = g->dt / ((float) g->options.nsteps);
	float equal_log_w = -logf((float) np);
	int n_newton_iterations = g->options.n_newton_iterations;

	hipMemcpy(&h_neff, g->d.neff, sizeof(float), hipMemcpyDeviceToHost); //neff for t = 0 should already be set
	resamplenow = h_neff < g->options.resamplethreshold; //do we need resampling now?
	if (!resamplenow && g->options.computenmean)
		initancestors_noresample<<<g->options.nblocks, NT>>>(g->d.ancestor + np, np);
	if (TIMECALC) {
		clock_gettime(CLOCK_REALTIME, &t0);
		tsum.tv_sec = 0; tsum.tv_nsec = 0;
	}
	for (int t = 1; t < g->T; t++) {

		offset_ancestors = (t & g->d.maxK) * np; //ancestors are always circularly buffered with K+1 columns so we can do fast modular arithmetic in calc_moments
		offset_Kbuffers = (t % (g->options.K + 1)) * np; //gp doesn't need to have a power of 2 number of columns since we don't need modular arithmetic on the GPU for it
		offset_ns = offset_Kbuffers * g->options.nsteps; //ns is circularly buffered with a number of columns that may not be a power of 2 to save memory since we don't need fast modular arithmetic to index its columns inside a GPU kernel.

		if (resamplenow) {
			nresamples++;
			prevoffset = offset;
			offset = np - offset;
			hipcub::DeviceScan::InclusiveSum(g->d.d_temp_storage_float_add_scan, g->d.temp_storage_bytes_float_add_scan, g->d.w, g->d.W, np);
			W2O<<<g->options.nblocks, BLOCKSIZE>>>(g->h.u[t], np, g->d.W, g->d.O);
			hipcub::DeviceScan::InclusiveScan(g->d.d_temp_storage_uint_max_scan, g->d.temp_storage_bytes_uint_max_scan,  g->d.O, g->d.Oi, hipcub::Max(), np);

			resample_shared<<<g->options.nblocks_rs, BLOCKSIZE_RS>>>(g->d.Oi, np, prevoffset, offset, (float4 *) g->d.states, (float4 *) g->d.cbr,
					g->d.log_w, g->d.log_w_corrected, g->d.ancestor + offset_ancestors);

			//need to normalize corrected weights to get back to a discrete probability distribution:
			hipcub::DeviceReduce::Max(g->d.d_temp_storage_float_max_reduction, g->d.temp_storage_bytes_float_max_reduction, g->d.log_w_corrected + offset, pd_mlw_corrected, np); //calculate max log weight
			subtract_and_exponentiate_correctedonly<<<g->options.nblocks, BLOCKSIZE>>>(g->d.log_w_corrected + offset, g->d.w_corrected, np); //subtract log of max weight and exponentiate
			hipcub::DeviceReduce::Sum(g->d.d_temp_storage_float_add_reduction, g->d.temp_storage_bytes_float_add_reduction, g->d.w_corrected, pd_sw_corrected,  np); //starting from raw corrected weights after resampling, we've now calculated their sum divided by their maximum
			//at this point, we haven't actually normalized the weights but we have the normalization constant which we'll use in later kernel calls
		}

		if (TIMECALC)
			clock_gettime(CLOCK_REALTIME, &ts);

		//advance the particle filter by one time step (i.e. nsteps substeps)
		advancepf<<<g->options.nblocks_pf, BLOCKSIZE_PF>>>(g->h.fobs[t] - foffset, g->options.nsteps, stepsize, n_newton_iterations,
				((float4 *) g->d.states) + offset, ((float4 *) g->d.cbr) + offset, g->d.gp + offset_Kbuffers,
				g->d.ns + offset_ns, //offset into spiking matrix that will be used to compute moments
				g->d.log_w, g->d.log_w_corrected + offset,
				g->d.q_spike + (t + g->options.ntimepoints_pre) * g->options.nsteps, (RTYPE *) g->d.rngstates, np, resamplenow, equal_log_w);

		//normalize weights, calculate their sums, etc.
		process_weights(g, t, offset);
		hipMemcpyFromSymbol(&h_neff, HIP_SYMBOL(d_neff), sizeof(float));

		if (TIMECALC) {
			clock_gettime(CLOCK_REALTIME, &ts2);
			td = tdiff(ts, ts2);
			tsum.tv_sec += td.tv_sec; tsum.tv_nsec += td.tv_nsec;
		}

	}

	if (TIMECALC) {
		clock_gettime(CLOCK_REALTIME, &t1);
		td = tdiff(t0, t1);
		std::cerr << "resampled on " << nresamples << " / " << g->T << " time steps\n";
		std::cerr << "total time: " << ((float) td.tv_sec + ((float) td.tv_nsec ) / 1000000000.0) << " s, ";
		std::cerr << "excluding resampling: " << ((float) tsum.tv_sec + ((float) tsum.tv_nsec ) / 1000000000.0) << " s\n";
	}

	return 1;
}

//this function numerically approximates the rate equation while randomly generating APs according to the particle filter's proposal distribution
//a backward (implicit) Euler method is used
__device__ float SimulateKinetics(float4 * __restrict__ pstates, float4 * __restrict__ pcbr,
		bool * __restrict__ ns,
		const float * __restrict__ q_spike,
		int n_newton_iterations,
		float stepsize,
		int nsteps, int np, int ii, //need np for the stride when assigning ns
		RTYPE *prngstate) {

	// load data:
	float s1prev = pstates->w;
	float s2prev = pstates->x;
	float s3prev = pstates->y;
	float s4prev = pstates->z;
	float s0prev = S - (s1prev + s2prev + s3prev + s4prev);  // calcium-free state is not explicitly stored
	float cprev = pcbr->w;
	float b0prev = pcbr->x;
	float b1prev = pcbr->y;

	float pq_spiking = 1.0;

	//local variables:
	int iter, jj;
	float c, a, denom, M_c_c, z_c;
	float rate0, rate1, rate2, rate3;
	float brate0, brate1;

	float s0, s1, s2, s3, s4;
	float M_c_s0, M_c_s1, M_c_s2, M_c_s3, M_c_s4;
    float M_s_c0, M_s_c1, M_s_c2, M_s_c3, M_s_c4;
    float M_s_s_diag0, M_s_s_diag1, M_s_s_diag2, M_s_s_diag3, M_s_s_diag4;
    float M_s_s_diagp10, M_s_s_diagp11, M_s_s_diagp12, M_s_s_diagp13;
    float z_s0, z_s1, z_s2, z_s3, z_s4;

    float b0, b1;
    float M_c_b0, M_c_b1;
    float M_b_c0, M_b_c1;
    float M_b_b0, M_b_b1;
    float z_b0, z_b1;
    float c_leak;

    if (USEKDEX) {
    	c_leak = maxex * c0 / (c0 + kd_ex); /* inward leak current */
    }

	for (jj = 0; jj < nsteps; jj++) {

		//sample spikes
		float next_q = q_spike[jj];
		bool spikenow = hiprand_uniform(prngstate) < next_q;
		if (spikenow) {

			pq_spiking *= (p_spike / next_q);
			cprev += 1.0;

		} else {

			pq_spiking *= ((1.0 - p_spike) / (1.0 - next_q));

		}
		ns[jj * np + ii] = spikenow;

		// initialize based on previous states (could we do better some other way)?
		c = cprev;
		s0 = s0prev; s1 = s1prev; s2 = s2prev; s3 = s3prev; s4 = s4prev;
		b0 = b0prev; b1 = b1prev;

		for (iter = 0; iter < n_newton_iterations; iter++) {

			if (USEKDEX) {
				M_c_c = 1.0 + stepsize * (maxex * kd_ex / ((c + kd_ex) * (c + kd_ex)) + kon0 * s0 + kon1 * s1 + kon2 * s2 + kon3 * s3 + kon_B0 * (Btot0 - b0) + kon_B1 * (Btot1 - b1));
			} else {
				M_c_c = 1.0 + stepsize * (maxex + kon0 * s0 + kon1 * s1 + kon2 * s2 + kon3 * s3 + kon_B0 * (Btot0 - b0) + kon_B1 * (Btot1 - b1));
			}

			rate0 = kon0 * c * s0 - koff0 * s1;
			rate1 = kon1 * c * s1 - koff1 * s2;
			rate2 = kon2 * c * s2 - koff2 * s3;
			rate3 = kon3 * c * s3 - koff3 * s4;

			z_s0 = s0prev - s0 + stepsize * (-rate0);
			z_s1 = s1prev - s1 + stepsize * (rate0 - rate1);
			z_s2 = s2prev - s2 + stepsize * (rate1 - rate2);
			z_s3 = s3prev - s3 + stepsize * (rate2 - rate3);
			z_s4 = s4prev - s4 + stepsize * (rate3);

			brate0 = kon_B0 * c * (Btot0 - b0) - koff_B0 * b0;
			brate1 = kon_B1 * c * (Btot1 - b1) - koff_B1 * b1;

			z_b0 = b0prev - b0 + stepsize * (brate0);
			z_b1 = b1prev - b1 + stepsize * (brate1);

			if (USEKDEX) {
				z_c = cprev - c - stepsize * (maxex * c / (c + kd_ex) - c_leak + rate0 + rate1 + rate2 + rate3 + brate0 + brate1);
			} else {
				z_c = cprev - c - stepsize * ((c - c0) * maxex + rate0 + rate1 + rate2 + rate3 + brate0 + brate1);
			}

			M_s_c0 = stepsize * kon0 * s0;
			M_s_c1 = stepsize * (kon1 * s1 - kon0 * s0);
			M_s_c2 = stepsize * (kon2 * s2 - kon1 * s1);
			M_s_c3 = stepsize * (kon3 * s3 - kon2 * s2);
			M_s_c4 = stepsize * (-kon3 * s3);

			M_c_s0 = stepsize * kon0 * c;
			M_c_s1 = stepsize * (kon1 * c - koff0);
			M_c_s2 = stepsize * (kon2 * c - koff1);
			M_c_s3 = stepsize * (kon3 * c - koff2);
			M_c_s4 = stepsize * (-koff3);

			M_s_s_diag0 = 1.0 + stepsize * kon0 * c;
			M_s_s_diag1 = 1.0 + stepsize * (kon1 * c + koff0);
			M_s_s_diag2 = 1.0 + stepsize * (kon2 * c + koff1);
			M_s_s_diag3 = 1.0 + stepsize * (kon3 * c + koff2);
			M_s_s_diag4 = 1.0 + stepsize * (koff3);

			M_s_s_diagp10 = -stepsize * koff0;
			M_s_s_diagp11 = -stepsize * koff1;
			M_s_s_diagp12 = -stepsize * koff2;
			M_s_s_diagp13 = -stepsize * koff3;

			M_c_b0 = -stepsize * (kon_B0 * c + koff_B0);
			M_c_b1 = -stepsize * (kon_B1 * c + koff_B1);

			M_b_c0 = -stepsize * kon_B0 * (Btot0 - b0);
			M_b_c1 = -stepsize * kon_B1 * (Btot1 - b1);

			M_b_b0 = 1.0 - M_c_b0;
			M_b_b1 = 1.0 - M_c_b1;

			/* solve linear system by Gaussian elimination */

			/* forward pass sets below-diagonal to zero and diagonal to one, except first row and column which change but remain dense */
			M_s_s_diagp10 /= M_s_s_diag0;
			z_s0          /= M_s_s_diag0;
			M_s_c0        /= M_s_s_diag0;

			a = -stepsize * kon0 * c; // first below diagonal element of M
			denom = M_s_s_diag1 - a * M_s_s_diagp10;
			z_s1   = (z_s1    - a *   z_s0) / denom;
			M_s_c1 = (M_s_c1  - a * M_s_c0) / denom;
			M_s_s_diagp11 /= denom;

			a = -stepsize * kon1 * c; // second below diagonal element of M
			denom = M_s_s_diag2 - a * M_s_s_diagp11;
			z_s2   = (z_s2    - a *   z_s1) / denom;
			M_s_c2 = (M_s_c2  - a * M_s_c1) / denom;
			M_s_s_diagp12 /= denom;

			a = -stepsize * kon2 * c; // third below diagonal element of M
			denom = M_s_s_diag3 - a * M_s_s_diagp12;
			z_s3   = (z_s3    - a *   z_s2) / denom;
			M_s_c3 = (M_s_c3  - a * M_s_c2) / denom;
			M_s_s_diagp13 /= denom;

			a = -stepsize * kon3 * c; // fourth below diagonal element of M
			denom = M_s_s_diag4 - a * M_s_s_diagp13;
			z_s4   = (z_s4    - a *   z_s3) / denom;
			M_s_c4 = (M_s_c4  - a * M_s_c3) / denom;

			/* backward pass sets above-diagonal and M_c_s to zero */
			z_s3   -= M_s_s_diagp13 * z_s4;
			M_s_c3 -= M_s_s_diagp13 * M_s_c4;
			z_c   -= M_c_s4 * z_s4;
			M_c_c -= M_c_s4 * M_s_c4;

			z_s2   -= M_s_s_diagp12 * z_s3;
			M_s_c2 -= M_s_s_diagp12 * M_s_c3;
			z_c   -= M_c_s3 * z_s3;
			M_c_c -= M_c_s3 * M_s_c3;

			z_s1   -= M_s_s_diagp11 * z_s2;
			M_s_c1 -= M_s_s_diagp11 * M_s_c2;
			z_c   -= M_c_s2 * z_s2;
			M_c_c -= M_c_s2 * M_s_c2;

			z_s0   -= M_s_s_diagp10 * z_s1;
			M_s_c0 -= M_s_s_diagp10 * M_s_c1;
			z_c   -= M_c_s1 * z_s1;
			M_c_c -= M_c_s1 * M_s_c1;

			z_c    -= M_c_s0 * z_s0;
			M_c_c  -= M_c_s0 * M_s_c0;

			/* forward pass only for buffers, sets M_b_b to one M_c_b to zero */
			z_b0   /= M_b_b0;
			M_b_c0 /= M_b_b0;
			z_c   -= M_c_b0 *   z_b0;
			M_c_c -= M_c_b0 * M_b_c0;

			z_b1   /= M_b_b1;
			M_b_c1 /= M_b_b1;
			z_c   -= M_c_b1 *   z_b1;
			M_c_c -= M_c_b1 * M_b_c1;

			z_c /= M_c_c; /* set M_c_c to one */

			/* forward pass eliminates M_s_c */
			z_s0 -= M_s_c0 * z_c;
			z_s1 -= M_s_c1 * z_c;
			z_s2 -= M_s_c2 * z_c;
			z_s3 -= M_s_c3 * z_c;
			z_s4 -= M_s_c4 * z_c;

			/* forward pass eliminates M_b_c */
			z_b0 -= M_b_c0 * z_c;
			z_b1 -= M_b_c1 * z_c;

			/* we've now reduced M to the identity matrix, so \Delta[c; s; b] = [z_c; z_s; z_b] */
			c  += z_c;
			s0 += z_s0;
			s1 += z_s1;
			s2 += z_s2;
			s3 += z_s3;
			s4 += z_s4;
			b0 += z_b0;
			b1 += z_b1;

		}

		// update states
		cprev = c;
		s0prev = s0; s1prev = s1; s2prev = s2; s3prev = s3; s4prev = s4;
		b0prev = b0; b1prev = b1;

	}

	pstates->w = s1prev;
	pstates->x = s2prev;
	pstates->y = s3prev;
	pstates->z = s4prev;

	pcbr->w = cprev;
	pcbr->x = b0prev;
	pcbr->y = b1prev;

	return pq_spiking;

}


//this kernel advances states and weights one time step in the particle filter
__global__ void advancepf(float fcorr, int nsteps, float stepsize, int n_newton_iterations,
		float4 *states_all, float4 *cbr_all, float * __restrict__ gp,
		bool * __restrict__ ns,
		float * __restrict__ log_wall, float * __restrict__ log_w_correctedall,
		const float * __restrict__ q_spike, RTYPE *rngstates, int np, bool resamplenow, float equal_log_w) {
	int tid = threadIdx.x;
	int ii_base = tid + blockIdx.x * BLOCKSIZE_PF;
	float4 states, cbr;
	float pq_spiking, vFtotal, log_pobs, nextgpval;
	RTYPE rngstate = rngstates[ii_base];
	int ii = ii_base;
	__shared__ float log_w_corrected_offset;

	if (!threadIdx.x) {
		if (resamplenow) //to get normalized weights, we divide by max(w) * sum(w / max(w))
			log_w_corrected_offset = max_log_w_corrected + logfunc(sum_w_corrected);
		else
			log_w_corrected_offset = 0.0;
	}

	__syncthreads();

	while (ii < np) {

		states = states_all[ii];
		cbr = cbr_all[ii];

		pq_spiking = SimulateKinetics(&states, &cbr, ns, q_spike, n_newton_iterations, stepsize, nsteps, np, ii, &rngstate);
		cbr.z += hiprand_normal(&rngstate) * sigr_sqrtdt; // baseline drift

		states_all[ii] = states;
		cbr_all[ii] = cbr;

		nextgpval = expfunc(cbr.z) * (FBGp1 + (states.w * db1 + states.x * db2 + states.y * db3 + states.z * db4) / S);  // gain * photon flux

		gp[ii] = nextgpval;

		vFtotal = basevar + gain * nextgpval;
		log_pobs = -0.5 * ((nextgpval - fcorr) * (nextgpval - fcorr) / vFtotal + logfunc(((float) (2.0 * M_PI)) * vFtotal));

		if (resamplenow)
			log_wall[ii] = equal_log_w + log_pobs;
		else
			log_wall[ii] += log_pobs;
		log_w_correctedall[ii] += (log_pobs + logfunc(pq_spiking) - log_w_corrected_offset);
		ii += BLOCKSIZE_PF * gridDim.x;

	}
	rngstates[ii_base] = rngstate;
	if (!ii_base) { //reset weight maxima to save two cudamemcpytosymbol calls
		mlw_total = INT_MIN;
		mlw_c_total = INT_MIN;
	}
}


__global__ void InitializePF(
		float fcorr, float fcorr_approx, int nsteps, float stepsize, int n_newton_itrations, //fcorr is observed fluorescence - fdc
		float4 *states_all, float4 *cbr_all, float * gp,
		bool *ns_pre, bool *ns,
		float *log_wall, float *log_w_correctedall,
		const float * __restrict__ q_spike, RTYPE *rngstates,
		int np, int ntimepoints_pre,
		float4 states0, float4 cbr0,
		float mu_r_init, float var_r_init, float vFtotal_init) {

	int ii_base = threadIdx.x + blockIdx.x * BLOCKSIZE_PF;
	RTYPE rngstate = rngstates[ii_base];
	int ii = ii_base;
	float pq_spiking, b, rmaxexp, rmax, dfac, likvarr, likmeanr, v_r, m_r, r, log_priorp_r, log_samplingp_r, expr, vFtotal, log_pobs, log_w_total;

	float4 states, cbr;

	while (ii < np) {

		states = states0;
		cbr = cbr0;

		pq_spiking  = SimulateKinetics(&states, &cbr, ns_pre, q_spike, 							  n_newton_itrations, stepsize, nsteps * ntimepoints_pre, np, ii, &rngstate); //simulate up to one full dt before first measurement
		pq_spiking *= SimulateKinetics(&states, &cbr, ns,     q_spike + nsteps * ntimepoints_pre, n_newton_itrations, stepsize, nsteps,                   np, ii, &rngstate); //simulate up to time of first measurement
		states_all[ii] = states;

		//we now do conditional sampling of r given s and f
		b = FBGp1 + (states.w * db1 + states.x * db2 + states.y * db3 + states.z * db4) / S;

		//do a linear approximation of the exponential function at rmax:
		rmaxexp = fcorr_approx / b; //always positive
		rmax = logfunc(rmaxexp); //r with maximum likelihood given s, F - fdc =  fcorr_approx. this is where we'll do the linearization of exp(r)
		dfac = rmaxexp * b; //denominator factor for calculating a Gaussian approximation to P[F | r]. since rmax has been chosen to match F(1), this is usually F(1) - P.fdc
		likvarr = vFtotal_init / (dfac * dfac); //variance of a Gaussian approximation of likelihood P[F | r]
		likmeanr = fcorr / dfac + rmax - 1.0; //mean of a Gaussian approximation of likelihood P[F | r]. note that unlike fcorr_approx, fcorr can be negative

		v_r = 1.0 / (1.0 / likvarr + 1.0 / var_r_init);
		m_r = v_r * (likmeanr / likvarr + mu_r_init / var_r_init);

		r = m_r + sqrt(v_r) * hiprand_normal(&rngstate);
		log_priorp_r    = -0.5 * ((r - mu_r_init) * (r - mu_r_init) / var_r_init + logfunc(((float) (2.0 * M_PI)) * var_r_init));
		log_samplingp_r = -0.5 * ((r - m_r)       * (r - m_r)       / v_r        + logfunc(((float) (2.0 * M_PI)) * v_r));
		expr = expfunc(r);
		gp[ii] = expr * b; //gain * photon flux
		vFtotal = basevar + gain * gp[ii];

		log_pobs = -0.5 * ((gp[ii] - fcorr) * (gp[ii] - fcorr) / vFtotal + logfunc(((float) (2.0 * M_PI)) * vFtotal));
		log_w_total = log_pobs + log_priorp_r - log_samplingp_r;
		cbr.z = r;
		cbr_all[ii] = cbr;

		log_wall[ii] = log_w_total;
		log_w_correctedall[ii] = log_w_total + logfunc(pq_spiking);
		ii += BLOCKSIZE_PF * gridDim.x;

	}
	rngstates[ii_base] = rngstate;
	if (!ii_base) { //reset weight maxima to save two cudamemcpytosymbol calls
		mlw_total = INT_MIN;
		mlw_c_total = INT_MIN;
	}

}


void gpu5s_initialstates(gpu5s_problem *g) {
	//calculate the equilibrium solution to the rate equations at resting calcium:
	float Ka;
	float4 states0, cbr0;
	float c0 = g->params.c0;
	float S = g->params.S;
	float Btot0 = g->params.Btot0;
	float Btot1 = g->params.Btot1;

	cbr0.w = c0;
	if (c0 == 0.0) {

		states0.w = 0.0; states0.x = 0.0; states0.y = 0.0; states0.z = 0.0;
		cbr0.x = 0.0; cbr0.y = 0.0;

	} else {

		float v1 = 	    c0 * (g->params.kon0 / g->params.koff0);
		float v2 = v1 * c0 * (g->params.kon1 / g->params.koff1);
		float v3 = v2 * c0 * (g->params.kon2 / g->params.koff2);
		float v4 = v3 * c0 * (g->params.kon3 / g->params.koff3);
		float P = 1.0 + v1 + v2 + v3 + v4;
		states0.w = S * v1 / P;
		states0.x = S * v2 / P;
		states0.y = S * v3 / P;
		states0.z = S * v4 / P;

		Ka = g->params.kon_B0 / g->params.koff_B0;
		cbr0.x = Btot0 * c0 * Ka / (1.0 + c0 * Ka);
		Ka = g->params.kon_B1 / g->params.koff_B1;
		cbr0.y = Btot1 * c0 * Ka / (1.0 + c0 * Ka);

	}
	//std::cerr << "x0 " << x0 << " s0 " << s0 << " c0 " <<  g->params.c0 << "\n";

	float fmin = fabs(g->h.fobs[0] * 0.05); //minimum value of f from which we would initialize r etc., regardless of the value of fdc. FIXME kind of a hack
	float fcorr = g->h.fobs[0] - g->params.fdc; //can be negative due to noise, wrong parameters, etc.
	float fcorr_approx; //value of corrected (by fdc) fluorescence we'll use to determine the prior on r, linearize the exponential function, and calculate the laplace approximation of the observation function
	if (fcorr < fmin) {
		std::cerr << "Warning: initial corrected fluorescence is too low for standard initialization technique, using 5% of first uncorrected fluorescence value\n";
		fcorr_approx = fmin;
	} else {
		fcorr_approx = 0.95 * fcorr;
	}

	float beq = g->params.FBGp1 + (states0.w * g->params.db1 + states0.x * g->params.db2 + states0.y * g->params.db3 + states0.z * g->params.db4) / g->params.S;
	float mu_r_init = logf(fcorr_approx / beq);
	float var_r_init = log(R_INIT_SD) * log(R_INIT_SD); //prior variance on r, FIXME defs and units. also for 3s? why is this passed to the kernel? should be pure preprocessor defs.

	float vFtotal_init = g->params.vF + g->params.gain * fcorr_approx;
	float stepsize = g->dt / ((float) g->options.nsteps);
	unsigned int uz = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(retirementCount),     &uz, sizeof(unsigned int));

	InitializePF<<<g->options.nblocks_pf, BLOCKSIZE_PF>>>(
			fcorr, fcorr_approx, g->options.nsteps, stepsize, g->options.n_newton_iterations,
			(float4 *) g->d.states, (float4 *) g->d.cbr, g->d.gp,
			g->d.ns_pre, g->d.ns,
			g->d.log_w, g->d.log_w_corrected,
			g->d.q_spike, (RTYPE *) g->d.rngstates,
			g->options.nparticles, g->options.ntimepoints_pre,
			states0, cbr0,
			mu_r_init, var_r_init, vFtotal_init);

	process_weights(g, 0, 0);
}


__global__ void resample_shared(const int * __restrict__ O, int np, int prevoffset, int offset,
		float4 *states_all, float4 *cbr_all, float *log_w, float *log_w_corrected, int *ancestor) {
	//this version of the resampling kernel uses shared memory to spread work out equally among threads, so we don't have to wait a long time for a single thread to do a lot of work.

	//note that we don't reorder the elements ns or or gp, but instead keep track of ancestors for them so we can calculate moments later
	int tid = threadIdx.x;
	int ii = threadIdx.x + blockIdx.x * BLOCKSIZE_RS;
	int prevO, jj, threadO, parent;
	float4 states_new, cbr_new;
	float lwcnew;
	__shared__ int sO[BLOCKSIZE_RS];
	__shared__ int blockwise_ancestor_index[BLOCKSIZE_RS];
	__shared__ int blockprevO, blockO;
	while (ii < np) {
		if (!ii) {
			prevO = 0;
		} else {
			prevO = O[ii - 1];
		}
		threadO = O[ii];
		sO[tid] = threadO; //copy particle counts to shared memory
		if (!tid) {
			blockprevO = prevO;
		} else if (tid == BLOCKSIZE_RS - 1) {
			blockO = threadO;
		}
		__syncthreads(); //broadcast updates to shared values to other threads
		int nvals_per_thread = (blockO - blockprevO + BLOCKSIZE_RS - 1) / BLOCKSIZE_RS; //divide blockwide number of values by number of threads in the block, and round up
		//we are now going to build a list of initial ancestors indices for each thread in the block.
		//each thread will write up to nvals_per_thread ancestors total.
		for (jj = (prevO - blockprevO + nvals_per_thread - 1) / nvals_per_thread; jj * nvals_per_thread < threadO - blockprevO; jj++)
			blockwise_ancestor_index[jj] = tid; //identify all threads which should start with a given blockwise ancestor index. some ancestors with low weights may not have any threads to give them child particles
		__syncthreads(); //broadcast updates to shared values to other threads
		int thread_offset = blockprevO + tid * nvals_per_thread; //initial offset into OUTPUT particles for this thread
		int thread_offset_max = MIN(blockprevO + (tid + 1) * nvals_per_thread, blockO); //final offset into OUTPUT particles for this thread, plus one
		int thread_blockwise_ancestor = blockwise_ancestor_index[tid]; //initial offset into this block's INPUT particles for this thread
		while (thread_offset < thread_offset_max) {
			parent = blockIdx.x * BLOCKSIZE_RS + thread_blockwise_ancestor;
			states_new = states_all[parent + prevoffset];
			cbr_new = cbr_all[parent + prevoffset];

			lwcnew = log_w_corrected[parent + prevoffset] - log_w[parent]; //log_w is not double buffered
			while (thread_offset < thread_offset_max && thread_offset < sO[thread_blockwise_ancestor]) {

				states_all[thread_offset + offset] = states_new;
				cbr_all[thread_offset + offset] = cbr_new;

				log_w_corrected[thread_offset + offset] = lwcnew; //log_w is not double buffered
				ancestor[thread_offset] = parent; //FIXME don't do this if we're not calculating any moments
				thread_offset++; //move on to the next value

			}
			while (thread_offset < thread_offset_max && thread_offset >= sO[thread_blockwise_ancestor]) //find the next non empty bin if we've already exhausted the present bin and haven't yet assigned all this thread's values.
				thread_blockwise_ancestor++;
		}
		ii += BLOCKSIZE_RS * gridDim.x; //grid stride loops
	}
	//FIXME now we need to do a reduction on the corrected weights, currently this is in another kernel
}


__device__ float floataddblockreduce_fromscalar(float x, float *sdata, int tid) {
	const int SecSize = NT / WARPSIZE;
	int lane = (SecSize - 1) & tid;    //lane within a section, not within a warp
	int sec = tid / SecSize;
	#pragma unroll
	for (int offset = 1; offset < SecSize; offset *= 2)  //values are further reduced within a section
		x += __shfl_down(x, offset, SecSize);
	if (!lane)
		sdata[sec] = x; //write each section's reduction into shared memory
	__syncthreads();
	if(tid < WARPSIZE) { //we now reduce the remaining values into a single block reduction, using a single warp
		x = sdata[tid];
	#pragma unroll
		for(int offset = 1; offset < WARPSIZE; offset *= 2) {
			x += __shfl_down(x, offset);
		}
		sdata[tid] = x;
	}
	__syncthreads();
	float blockreduction = sdata[0];
	//__syncthreads(); this line is present in MGPU, not really clear on why
	return blockreduction;
}

__device__ float floataddblockreduce(float *rdata, float *sdata, int tid) {
	float x = rdata[0];
	#pragma unroll
	for (int i = 1; i < VT; i++)
		x += rdata[i]; //each thread reduces VT values sequentially
	return floataddblockreduce_fromscalar(x, sdata, tid);
}

__device__ float floatmaxblockreduce_fromscalar(float x, float *sdata, int tid) {
	float xn;
	const int SecSize = NT / WARPSIZE;
	int lane = (SecSize - 1) & tid;    //lane within a section, not within a warp
	int sec = tid / SecSize;
	#pragma unroll
	for (int offset = 1; offset < SecSize; offset *= 2) { //values are further reduced within a section
		xn = __shfl_down(x, offset, SecSize);
		x = MAX(x, xn);
	}
	if (!lane)
		sdata[sec] = x; //write each section's reduction into shared memory
	__syncthreads();
	if(tid < WARPSIZE) { //we now reduce the remaining values into a single block reduction, using a single warp
		x = sdata[tid];
	#pragma unroll
		for(int offset = 1; offset < WARPSIZE; offset *= 2) {
			xn = __shfl_down(x, offset);
			x = MAX(x, xn);
		}
		sdata[tid] = x;
	}
	__syncthreads();
	float blockreduction = sdata[0];
	//__syncthreads(); this line is present in MGPU, not really clear on why
	return blockreduction;
}

__device__ float floatmaxblockreduce(float *rdata, float *sdata, int tid) {
	float x = rdata[0];
	#pragma unroll
	for (int i = 1; i < VT; i++)
		x = MAX(x, rdata[i]); //each thread reduces VT values sequentially
	x = floatmaxblockreduce_fromscalar(x, sdata, tid);
	return x;
}

__device__ void global2reg(float *g, float *r, int tid, int count, float initval) { //copies data from device global memory to per-thread register array
	if (count >= NT * VT) { //FIXME >= NT * VT - 1 ????
	#pragma unroll
		for (int i = 0; i < VT; i++)
			r[i] = g[NT * i + tid];
	} else {
	#pragma unroll
		for (int i = 0; i < VT; i++) {
			int index = NT * i + tid;
			r[i] = initval;
			if (index < count) r[i] = g[index];
		}
	}
}

//FIXME should template bool args
__device__ float log2linblocksum(int tid, float * sdata, float x0, float *r, float *g_x, float &sumsq, int count, bool writelin, bool reducesquare) {
	float sum = 0.0;
	for (int i = 0; i < VT; i++) {
		if (NT * i + tid < count) {
			float v = expfunc(r[i] - x0);
			sum += v;
			if (writelin)
				g_x[NT * i + tid] = v;
			if (reducesquare)
				sumsq += v * v;
		}
	}
	__syncthreads();
	sum = floataddblockreduce_fromscalar(sum, sdata, tid);
	if (reducesquare) {
		__syncthreads();
		sumsq = floataddblockreduce_fromscalar(sumsq, sdata, tid);
	}
	return sum;
}


__inline__ __device__ void findlastblock(bool &amLast) { //amLast should be a shared bool variable
	__threadfence(); //make sure all previous atomic operations (e.g. from reductions) are flushed before we take a ticket
	if (!threadIdx.x)
	{
		unsigned int ticket = atomicInc(&retirementCount, gridDim.x);
		// If the ticket ID is equal to the number of blocks, we are the last block!
		amLast = (ticket == gridDim.x - 1);
		if (amLast)
			retirementCount = 0;
	}
	__syncthreads(); //so that all threads get the updated value of the share bool amLast
}


__global__ void reduce_weights(float *lw, float *lw_c, float *w, float *w_c, int np,
		float *block_lw_max, float *block_sw_over_max, float *block_swsq_over_maxsq,
		float *block_lw_c_max, float *block_sw_c_over_max,
		int t, float *log_sum_raw_w, float *neff, bool calcmoments) {
	//this function and its subfunctions accomplish several tasks:
	//1) calculates logs of sums of weights (corrected and uncorrected)
	//2) calculates neff = 1 / sum(w * w), where w has been normalized (uncorrected weights only)
	//3) populates array of linear uncorrected weights divided by max within their block (log weights are NOT adjusted by this function from their raw values!)
	//4) stores max of log corrected/uncorrected weights for each block (sums per block of weights divide by max within the block are also stored, but not later used)
	//based in part on MGPU's reduction algorithm (https://github.com/NVlabs/moderngpu) and the cuda sample "threadfencereduction"
	int tid = threadIdx.x;
	__shared__ float sdata[WARPSIZE];
	int block_rw_offset = blockIdx.x * NT * VT; //offset for this block when reading from or writing to an array of size np in global device memory
	float swsq = 0.0, swsq_c; //hopefully swsq_c is optimized out as nothing is done to it
	float rdata[VT];

	global2reg(lw + block_rw_offset, rdata, tid, np - block_rw_offset, -FLT_MAX);
	__syncthreads();
	float mlw = floatmaxblockreduce(rdata, sdata, tid); //max of log weights for this block
	float sw   = log2linblocksum(tid, sdata, mlw,   rdata, w   + block_rw_offset, swsq,   np - block_rw_offset, true,        true); //we need sum of uncorrected weights and their sum of squares for resampling

	global2reg(lw_c + block_rw_offset, rdata, tid, np - block_rw_offset, -FLT_MAX);
	__syncthreads();
	float mlw_c = floatmaxblockreduce(rdata, sdata, tid); //max of log weights for this block
	float sw_c = log2linblocksum(tid, sdata, mlw_c, rdata, w_c + block_rw_offset, swsq_c, np - block_rw_offset, calcmoments, false); //we need corrected weights for moment calculation. we pass swsq_c but nothing should be done to it

	if (!tid) { //first thread of each block stores the results and calls atomic max functions
		block_lw_max[blockIdx.x] = mlw;
		block_sw_over_max[blockIdx.x] = sw;
		block_swsq_over_maxsq[blockIdx.x] = swsq;
		block_lw_c_max[blockIdx.x] = mlw_c;
		block_sw_c_over_max[blockIdx.x] = sw_c;
		//do an atomic max to store the truncated max logarithms of corrected/uncorrected weights weight
		atomicMax(&mlw_total, (int) mlw);
		atomicMax(&mlw_c_total, (int) mlw_c);
	}
	__shared__ bool amLast;
	findlastblock(amLast);
	if (amLast) { //the last block completes the reductions
		__shared__ float log_totalsw;
		float mlw_r = (float) mlw_total;
		float mlw_c_r = (float) mlw_c_total;
		float threadsw = 0.0;
		float threadsw_c = 0.0;
		for (int i = tid; i < gridDim.x; i += NT) { //whereas normally each thread reads VT values, here each thread reads nblocks / NT values
			threadsw   += expfunc(block_lw_max[i]   - mlw_r  ) * block_sw_over_max[i]; //each term is corrected so that it contributed the block's sum of weight divided by TOTAL max weight
			threadsw_c += expfunc(block_lw_c_max[i] - mlw_c_r) * block_sw_c_over_max[i];
		}
		__syncthreads();
		threadsw = floataddblockreduce_fromscalar(threadsw, sdata, tid); //ok if some threads haven't done anything in the above for loop since threadsw was zeroed.
		threadsw_c = floataddblockreduce_fromscalar(threadsw_c, sdata, tid); //ok if some threads haven't done anything in the above for loop since threadsw was zeroed.
		if (!tid)
			log_totalsw = logfunc(threadsw) + mlw_r; //log of sum of ALL weights
		__syncthreads();
		float log_totalsw_r = log_totalsw;
		float threadswsq = 0.0;
		for (int i = tid; i < gridDim.x; i += NT) {
			float qfac = expfunc(block_lw_max[i] - log_totalsw_r);
			threadswsq += qfac * qfac * block_swsq_over_maxsq[i];
		}
		__syncthreads();
		threadswsq = floataddblockreduce_fromscalar(threadswsq, sdata, tid); //ok if some threads haven't done anything in the above for loop since threadsw was zeroed.
		if(!tid) {
			log_total_sum_weights = log_totalsw_r;
			log_total_sum_weights_c = logfunc(threadsw_c) + mlw_c_r;
			log_sum_raw_w[t] = log_total_sum_weights_c;
			neff[t] = 1.0 / threadswsq;
			d_neff  = 1.0 / threadswsq;
		}
	}
}

//fixme template bool params
__global__ void normalize_weights(float *w, float *lw, float *w_c, float *lw_c, float *block_lw_max, float *block_lw_c_max, int np, bool resamplenow, bool calcmoments) {
	//we need to adjust weights, log weights and log corrected weights
	int tid = threadIdx.x;
	int block_rw_offset = blockIdx.x * NT * VT; //offset for this block when reading from or writing to an array of size np in global device memory
	float dlog = log_total_sum_weights; //calculate the necessary offset for log weights in this block
	float dlog_c = log_total_sum_weights_c; //calculate the necessary offset for log corrected weights in this block
	float dlin, dlin_c;
	if (calcmoments)
		dlin_c = expfunc(log_total_sum_weights_c - block_lw_c_max[blockIdx.x]); //we've already divided each weight by the block maximum
	if (resamplenow)
		dlin   = expfunc(log_total_sum_weights   - block_lw_max[blockIdx.x]); //we've already divided each weight by the block maximum
	if (np >= NT * VT + block_rw_offset) {
		for (int i = 0; i < VT; i++) {
			int index = block_rw_offset + NT * i + tid;
			lw[index] -= dlog;
			lw_c[index] -= dlog_c;
			if (calcmoments)
				w_c[index] /= dlin_c;
			if (resamplenow)
				w[index] /= dlin;
		}
	} else {
		for (int i = 0; i < VT; i++) {
			int index = block_rw_offset + NT * i + tid;
			if (index < np) {
				lw[index] -= dlog;
				lw_c[index] -= dlog_c;
				if (calcmoments)
					w_c[index] /= dlin_c;
				if (resamplenow)
					w[index] /= dlin;
			}
		}
	}
	if (threadIdx.x + blockIdx.x * BLOCKSIZE == 0) {
		//set max to 0 and sum to 1, to indicate that if we don't resample then no more normalization needs to be performed
		max_log_w_corrected = 0.0;
		sum_w_corrected = 1.0;
	}
}

//Fixme __restrict__ might help here?
__global__ void calc_moments(float *w_c, bool *ns, float *gp, int *ancestor,
		float *nmean, float *gpmean, float *gpsqmean,
		float *block_sw_spike, float *block_wgp, float * block_wgpsq,
		int t, int np, int maxK, int K, int nsteps, bool initnextancestors) {
	int ii_base = threadIdx.x + blockIdx.x * BLOCKSIZE;
	int ii = ii_base;
	int a;
	int a_gs[VT];
	int ancestor_offset = 0; //this lets the below code work for K = 0
	int tid = threadIdx.x;
	float w_gs[VT];
	float spikew;
	__shared__ float sdata[WARPSIZE];
	//store the ancestor K steps back for (up to) VT particles, going by grid strides
	for (int gs = 0; gs < VT; gs++) { //grid strides
		if (ii < np) {
			a = ii;
			for (int s = 0; s < K; s++) {
				ancestor_offset = ((t - s) & maxK) * np; //x & maxK == x % (maxK + 1) since maxK + 1 is a power of 2
				a = ancestor[ancestor_offset + a];
			}
			a_gs[gs] = a; //note this is a register array
			w_gs[gs] = w_c[ii]; //note this is a register array. we could in theory do the memory read only if spikes occured, not sure if that would speed things up.
		}
		ii += BLOCKSIZE * gridDim.x; //grid strides
	}

	//if the next time step doesn't require resampling we set ancestors in advance now and skip the resampling kernel entirely
	if (initnextancestors) {
		ancestor_offset = ((t + 1) & maxK) * np; //x & maxK == x % (maxK + 1) since maxK + 1 is a power of 2
		ii = ii_base;
		while (ii < np) {
			ancestor[ancestor_offset + ii] = ii; //note that when K==maxK, the next time step is the same as K time steps back. but that's ok since this function only reads spikes K time steps back, not ancestors.
			ii += BLOCKSIZE * gridDim.x;
		}
	}

	//now that we've determined ancestors K time steps back, we calculate the moments
	//first, calculate reductions for moments with each thread block

	//spiking moments:
	for (int jj = 0; jj < nsteps; jj++) { //loop over substeps within the time point K full steps back
		ii = ii_base;
		spikew = 0.0;
		for (int gs = 0; gs < VT; gs++) { //grid strides
			if (ii < np &&
					ns[jj * np + a_gs[gs]]) { //this second condition indicates a spike occurred
				spikew += w_gs[gs]; //add up weights of spiking particles over grid strides
			}
			ii += BLOCKSIZE * gridDim.x; //grid strides
		}
		__syncthreads(); //necessary?
		spikew = floataddblockreduce_fromscalar(spikew, sdata, tid); //add up the weights of all particles in this block that spiked for this substep
		if (!tid) //even if there we no spikes for this block, we still need to perform the global memory write to get the right answer from the reduction
			block_sw_spike[blockIdx.x + jj * gridDim.x] = spikew; //write the sum of weights with spikes for this block to global memory
	}

	if (gp != NULL) {
		//moments of gain * photon flux
		float wgp = 0.0, wgpsq = 0.0;
		ii = ii_base;
		for (int gs = 0; gs < VT; gs++) { //grid strides
			if (ii < np) {
				wgp   += w_gs[gs] * gp[a_gs[gs]];
				wgpsq += w_gs[gs] * gp[a_gs[gs]] * gp[a_gs[gs]];
			}
			ii += BLOCKSIZE * gridDim.x; //grid strides
		}
		__syncthreads(); //necessary?
		wgp   = floataddblockreduce_fromscalar(wgp, sdata, tid);
		//no __syncthreads() here, as there's one near the end of floataddblockreduce_fromscalar
		wgpsq = floataddblockreduce_fromscalar(wgpsq, sdata, tid);
		if (!tid) {
			block_wgp[blockIdx.x]   = wgp; //write the sum of weights with spikes for this block to global memory
			block_wgpsq[blockIdx.x] = wgpsq; //write the sum of weights with spikes for this block to global memory
		}
	}

	//if we're the last block, reduce over blocks
	__shared__ bool amLast;
	findlastblock(amLast);
	if (amLast) { //last block completes reduction over weights to calculate moments
		//reduction over blocks for spiking moments:
		for (int jj = 0; jj < nsteps; jj++) {
			float thread_sw_spike = 0.0;
			for (int kk = tid; kk < gridDim.x; kk += NT) //whereas normally each thread reads up to VT values, here each thread reads up to nblocks / NT values
				thread_sw_spike += block_sw_spike[kk + jj * gridDim.x];
			__syncthreads(); //necessary?
			thread_sw_spike = floataddblockreduce_fromscalar(thread_sw_spike, sdata, tid); //ok if some threads haven't done anything in the above for loop since thread_sw_spike was zeroed
			if (!tid)  //first thread of last block performs global device memory write
				nmean[jj] = thread_sw_spike; //sum of all weights for particles with spikes on this substep
		}

		if (gp != NULL) {
			//reduction over blocks for gain * photon flux
			float thread_wgp = 0.0, thread_wgpsq = 0.0;
			for (int kk = tid; kk < gridDim.x; kk += NT) { //whereas normally each thread reads up to VT values, here each thread reads up to nblocks / NT values
				thread_wgp   += block_wgp[kk];
				thread_wgpsq += block_wgpsq[kk];
			}
			__syncthreads(); //necessary?
			thread_wgp = floataddblockreduce_fromscalar(thread_wgp, sdata, tid);
			//no __syncthreads() here, as there's one near the end of floataddblockreduce_fromscalar
			thread_wgpsq = floataddblockreduce_fromscalar(thread_wgpsq, sdata, tid);
			if (!tid) { //first thread of last block performs global device memory write
				gpmean[0]   = thread_wgp; //sum of all weights for particles with spikes on this substep
				gpsqmean[0] = thread_wgpsq; //sum of all weights for particles with spikes on this substep
			}
		}
	}
}

void calc_moments_lastK(gpu5s_problem * g) {
	int t = g->T - 1;
	bool initnextancestors = false;
	for (int k = MIN(g->options.K - 1, g->T - 1); k >= 0; k--) {
		int np = g->options.nparticles;

		//get offsets into the arrays we're going to pass to calc_moments
		int gp_offset     = ((t - k) % (g->options.K + 1)) * np;
		int ns_offset     = ((t - k) % (g->options.K + 1)) * np  * g->options.nsteps;
		int gpmean_offset = t - k;
		int nmean_offset  = (t + g->options.ntimepoints_pre - k) * g->options.nsteps;

		calc_moments<<<g->options.nblocks, NT>>>(g->d.w_corrected, g->d.ns + ns_offset, g->d.gp + gp_offset, g->d.ancestor,
				g->d.nmean + nmean_offset, g->d.gpmean + gpmean_offset, g->d.gpsqmean + gpmean_offset,
				g->d.block_sw_spike, g->d.block_wgp, g->d.block_wgpsq,
				t, np, g->d.maxK, k, g->options.nsteps, initnextancestors);

	}
}


void process_weights(gpu5s_problem *g, int t, int offset) {
	int np = g->options.nparticles;
	reduce_weights<<<g->options.nblocks, NT>>>(g->d.log_w, g->d.log_w_corrected + offset, g->d.w, g->d.w_corrected, np,
			g->d.block_lw_max, g->d.block_sw_over_max, g->d.block_swsq_over_maxsq, g->d.block_lw_c_max, g->d.block_sw_c_over_max,
			t, g->d.log_sum_raw_w, g->d.neff, g->options.computenmean);
	hipMemcpyFromSymbol(&h_neff, HIP_SYMBOL(d_neff), sizeof(float));

	bool laststep = t == g->T - 1;
	resamplenow = (h_neff < g->options.resamplethreshold) && !laststep; //do we need resampling now?
	bool initnextancestors = !resamplenow && !laststep;
	normalize_weights<<<g->options.nblocks, NT>>>(g->d.w, g->d.log_w, g->d.w_corrected, g->d.log_w_corrected + offset, g->d.block_lw_max, g->d.block_lw_c_max, np, resamplenow, g->options.computenmean);
	if (g->options.computenmean) {
		if (t >= g->options.K) { //standard filter smoother K time points back

			//get offsets into the arrays we're going to pass to calc_moments
			int gp_offset     = ((t - g->options.K) % (g->options.K + 1)) * np;
			int gpmean_offset = t - g->options.K;
			int nmean_offset  = (t + g->options.ntimepoints_pre - g->options.K) * g->options.nsteps;
			int ns_offset     = ((t - g->options.K) % (g->options.K + 1)) * np * g->options.nsteps;

			calc_moments<<<g->options.nblocks, NT>>>(g->d.w_corrected, g->d.ns + ns_offset, g->d.gp + gp_offset, g->d.ancestor,
					g->d.nmean + nmean_offset, g->d.gpmean + gpmean_offset, g->d.gpsqmean + gpmean_offset,
					g->d.block_sw_spike, g->d.block_wgp, g->d.block_wgpsq,
					t, np, g->d.maxK, g->options.K, g->options.nsteps, initnextancestors);

			if (t == g->options.K) //filter smoother for states before the first observation's (full) time step
				calc_moments<<<g->options.nblocks, NT>>>(g->d.w_corrected, g->d.ns_pre,     NULL, g->d.ancestor,
						g->d.nmean, NULL, NULL,
						g->d.block_sw_spike_pre, NULL, NULL,
						t, np, g->d.maxK, g->options.K, g->options.nsteps * g->options.ntimepoints_pre, false);

			if (laststep)
				calc_moments_lastK(g); //calculate moments for the final K time steps, for which the filter smoother window will be less than K full time steps since we've run out of data

		} else if (initnextancestors)
			initancestors_noresample<<<g->options.nblocks, NT>>>(g->d.ancestor + ((t + 1) % (g->options.K + 1)) * np, np); //for t >= K this is done inside of calc_moments instead
	}
}


void pushparameterstodevice(gpu5s_problem *gpr) {
	//put parameter variables on device

	float prob_spike = gpr->params.lambda / ((float) gpr->options.nsteps);
	float sigr_sqrtdt_h = gpr->params.sigma_r * sqrtf(gpr->dt);
	hipMemcpyToSymbol(HIP_SYMBOL(sigr_sqrtdt), &sigr_sqrtdt_h, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(p_spike), &prob_spike, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(db1), &(gpr->params.db1), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(db2), &(gpr->params.db2), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(db3), &(gpr->params.db3), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(db4), &(gpr->params.db4), sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(kon0), &(gpr->params.kon0), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(kon1), &(gpr->params.kon1), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(kon2), &(gpr->params.kon2), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(kon3), &(gpr->params.kon3), sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(koff0), &(gpr->params.koff0), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(koff1), &(gpr->params.koff1), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(koff2), &(gpr->params.koff2), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(koff3), &(gpr->params.koff3), sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(kon_B0),  &(gpr->params.kon_B0), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(koff_B0), &(gpr->params.koff_B0), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(kon_B1),  &(gpr->params.kon_B1), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(koff_B1), &(gpr->params.koff_B1), sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(FBGp1),   &(gpr->params.FBGp1), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(basevar), &(gpr->params.vF), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(gain),    &(gpr->params.gain), sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(c0),      &(gpr->params.c0), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(maxex),   &(gpr->params.maxex), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(kd_ex),   &(gpr->params.kd_ex), sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(S),     &(gpr->params.S),     sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(Btot0), &(gpr->params.Btot0), sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(Btot1), &(gpr->params.Btot1), sizeof(float));

}

__global__ void setup_kernel(RTYPE *rngstates, int nrng, unsigned long long seedval)
{
	int id = threadIdx.x + blockIdx.x * BLOCKSIZE_PF;
	/* Each thread gets same seed, a different sequence number, no offset */
	if (id < nrng)
		hiprand_init(seedval, id, 0, &rngstates[id]);
}

void cudaseedrng(int nblocks_pf, void *rngstates, int nrng, unsigned long long seedval){
	setup_kernel<<<nblocks_pf, BLOCKSIZE_PF>>>((RTYPE *) rngstates, nrng, seedval);
}

int cudainitrng(int nblocks_pf, void *&rngstates, unsigned long long seedval) {
	int nrng = nblocks_pf * BLOCKSIZE_PF; //one rng per thread, regardless of how much data each thread processes on various kernels
	if (hipMalloc(&rngstates, nrng * sizeof(RTYPE)) != hipSuccess)
		return -100;
	cudaseedrng(nblocks_pf, rngstates, nrng, seedval);
	return 1;
}

timespec tdiff(timespec start, timespec end)
{
	timespec temp;
	if ((end.tv_nsec-start.tv_nsec)<0) {
		temp.tv_sec = end.tv_sec-start.tv_sec-1;
		temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
	} else {
		temp.tv_sec = end.tv_sec-start.tv_sec;
		temp.tv_nsec = end.tv_nsec-start.tv_nsec;
	}
	return temp;
}

int setgridsize(gpu5s_problem *g, int np) {
	if (np > g->d.maxnparticles)
		return -1;
	int nblocks = (np % NV) ? np / NV + 1 : np / NV;
	if (nblocks > g->d.maxnblocks)
		return -2;
	g->options.nparticles = np;
	g->options.nblocks = nblocks;
	g->options.nblocks_pf = (np % NV_PF) ? np / NV_PF + 1 : np / NV_PF;
	g->options.nblocks_rs = (np % NV_RS) ? np / NV_RS + 1 : np / NV_RS;
	return 1;
}

int allocate_gpupfdata(gpu5s_problem *g, int maxT, int np, int maxnsteps, int maxK, int maxKp1substeps,
		int maxtotalsubsteps, int maxpresubsteps, unsigned long long seedval) {
	if (g == NULL)
		return -1;
	if (maxK <= 0 || ((maxK + 1) & maxK ))
		return -2; //maxK + 1 must be positive and a power of 2 so we can do fast modular arithmetic using bitwise AND
	g->d.maxT = maxT;
	g->d.maxK = maxK;
	g->d.maxnsteps = maxnsteps;
	g->d.maxKp1substeps = maxKp1substeps;
	g->d.maxtotalsubsteps = maxtotalsubsteps;
	g->d.maxpresubsteps = maxpresubsteps;
	g->d.maxnparticles = np;
	g->d.maxnblocks = (np % NV) ? np / NV + 1 : np / NV;

	g->options.vt = VT; g->options.vt_pf = 1; g->options.vt_rs = VT_RS;
	g->options.nt = BLOCKSIZE; g->options.nt_pf = BLOCKSIZE_PF; g->options.nt_rs = BLOCKSIZE_RS;
	//initialize options to max array sizes
	if (setgridsize(g, np) < 0) {
		free_gpupfdata(g);
		return -3;
	}
	g->options.K = maxK;
	//set all CPU/GPU pointers to NULL
	g->h.fobs = NULL; g->h.u = NULL;
	g->d.rngstates = NULL;
	g->d.d_temp_storage_float_max_reduction = NULL; g->d.d_temp_storage_float_add_reduction = NULL;
	g->d.d_temp_storage_float_add_scan = NULL; g->d.d_temp_storage_uint_max_scan = NULL;
	g->d.states = NULL; g->d.cbr = NULL;
	g->d.ns = NULL; g->d.ns_pre = NULL;
	g->d.log_w = NULL; g->d.w = NULL; g->d.log_w_corrected = NULL; g->d.w_corrected = NULL; g->d.W = NULL;
	g->d.block_lw_max = NULL; g->d.block_sw_over_max = NULL; g->d.block_swsq_over_maxsq = NULL; g->d.block_lw_c_max = NULL; g->d.block_sw_c_over_max = NULL;
	g->d.block_sw_spike = NULL; g->d.block_sw_spike_pre = NULL; g->d.block_wgp = NULL; g->d.block_wgpsq = NULL;
	g->d.log_sum_raw_w = NULL; g->d.neff = NULL;
	g->d.q_spike = NULL; g->d.log_pq_spike = NULL; g->d.log_pq_nospike = NULL;
	g->d.nmean = NULL; g->d.gpmean = NULL; g->d.gpsqmean = NULL;
	g->d.O = NULL; g->d.Oi = NULL; g->d.ancestor = NULL;

	int npf  = np  			    * sizeof(float);
	int npi  = np   		    * sizeof(int);
	int Tf   = maxT 			* sizeof(float);
	int nsf  = maxtotalsubsteps * sizeof(float);
	int nbf  = g->d.maxnblocks  * sizeof(float);

	if (
			hipMalloc((void **) &(g->d.states), 2 * np * sizeof(float4))				!= hipSuccess ||
			hipMalloc((void **) &(g->d.cbr),    2 * np * sizeof(float4))				!= hipSuccess ||
			hipMalloc((void **) &(g->d.gp), 	npf * (maxK + 1)) 						!= hipSuccess ||
			hipMalloc((void **) &(g->d.ns),     maxKp1substeps * np * sizeof(bool)) 	!= hipSuccess ||
			hipMalloc((void **) &(g->d.ns_pre), maxpresubsteps * np * sizeof(bool))	!= hipSuccess ||
			hipMalloc((void **) &(g->d.log_w), npf) 									!= hipSuccess || //weights (only raw corrected weights are double buffered)
			hipMalloc((void **) &(g->d.w),     npf) 									!= hipSuccess ||
			hipMalloc((void **) &(g->d.log_w_corrected), 2 * npf) 						!= hipSuccess ||
			hipMalloc((void **) &(g->d.w_corrected),     npf) 							!= hipSuccess ||
			hipMalloc((void **) &(g->d.W), npf) 					 					!= hipSuccess ||
			hipMalloc((void **) &(g->d.block_lw_max), 			nbf) 					!= hipSuccess ||
			hipMalloc((void **) &(g->d.block_sw_over_max),		nbf) 					!= hipSuccess ||
			hipMalloc((void **) &(g->d.block_swsq_over_maxsq), nbf) 					!= hipSuccess ||
			hipMalloc((void **) &(g->d.block_lw_c_max), 		nbf) 					!= hipSuccess || //block reduction arrays
			hipMalloc((void **) &(g->d.block_sw_c_over_max), 	nbf) 					!= hipSuccess ||
			hipMalloc((void **) &(g->d.block_sw_spike),     	nbf * maxnsteps) 		!= hipSuccess ||
			hipMalloc((void **) &(g->d.block_sw_spike_pre), 	nbf * maxpresubsteps) 	!= hipSuccess ||
			hipMalloc((void **) &(g->d.block_wgp), 		 	nbf)			 		!= hipSuccess ||
			hipMalloc((void **) &(g->d.block_wgpsq), 		 	nbf)			 		!= hipSuccess ||
			hipMalloc((void **) &(g->d.log_sum_raw_w), Tf)  							!= hipSuccess || //T length arrays
			hipMalloc((void **) &(g->d.neff), Tf)           							!= hipSuccess ||
			hipMalloc((void **) &(g->d.q_spike), nsf) 									!= hipSuccess || //spiking proposal
			hipMalloc((void **) &(g->d.log_pq_spike), nsf) 							!= hipSuccess ||
			hipMalloc((void **) &(g->d.log_pq_nospike), nsf) 							!= hipSuccess ||
			hipMalloc((void **) &(g->d.nmean),   nsf) 									!= hipSuccess || //moments
			hipMalloc((void **) &(g->d.gpmean),   Tf) 									!= hipSuccess || //moments
			hipMalloc((void **) &(g->d.gpsqmean), Tf) 									!= hipSuccess || //moments
			hipMalloc((void **) &(g->d.O), npi)					   	  				!= hipSuccess ||
			hipMalloc((void **) &(g->d.Oi), npi)					     				!= hipSuccess ||
			hipMalloc((void **) &(g->d.ancestor), npi * (maxK + 1)) 					!= hipSuccess
	) {
		free_gpupfdata(g);
		return -4;
	}

	//FIXME have an option for a default seed where the rngstates are precomputed and compiled into the program
	if (cudainitrng(g->options.nblocks_pf, g->d.rngstates, seedval) < 0) { //always initialize with the same seed
		free_gpupfdata(g);
		return -5;
	}

	if (InitializeCubTemporaryArrays(g) < 0) {
		free_gpupfdata(g);
		return -6;
	}

	return 1;
}

void destroy_gpu5s_problem(gpu5s_problem *g) {
	free_gpupfdata(g);

	if (g->h.fobs != NULL)
		free(g->h.fobs);
	if (g->h.u != NULL)
		free(g->h.u);

	g->h.fobs = NULL; g->h.u = NULL;
	free(g);
}

//FIXME template these freeifnotnull functions
void cufreeifnotnull(float *&p) {
	if (p != NULL) {
		hipFree(p);
		p = NULL;
	}
}

void cufreeifnotnull(void *&p) {
	if (p != NULL) {
		hipFree(p);
		p = NULL;
	}
}

void cufreeifnotnull(bool *&p) {
	if (p != NULL) {
		hipFree(p);
		p = NULL;
	}
}

void cufreeifnotnull(int *&p) {
	if (p != NULL) {
		hipFree(p);
		p = NULL;
	}
}

void free_gpupfdata(gpu5s_problem *g) {
	if (g == NULL)
		return;

	cufreeifnotnull(g->d.rngstates);
	cufreeifnotnull(g->d.cbr);
	cufreeifnotnull(g->d.states);
	cufreeifnotnull(g->d.gp);
	cufreeifnotnull(g->d.ns);
	cufreeifnotnull(g->d.ns_pre);
	cufreeifnotnull(g->d.log_w);
	cufreeifnotnull(g->d.w);
	cufreeifnotnull(g->d.log_w_corrected);
	cufreeifnotnull(g->d.w_corrected);
	cufreeifnotnull(g->d.W);
	cufreeifnotnull(g->d.block_lw_max);
	cufreeifnotnull(g->d.block_sw_over_max);
	cufreeifnotnull(g->d.block_swsq_over_maxsq);
	cufreeifnotnull(g->d.block_lw_c_max);
	cufreeifnotnull(g->d.block_sw_c_over_max);
	cufreeifnotnull(g->d.block_sw_spike);
	cufreeifnotnull(g->d.block_sw_spike_pre);
	cufreeifnotnull(g->d.block_wgp);
	cufreeifnotnull(g->d.block_wgpsq);
	cufreeifnotnull(g->d.log_sum_raw_w);
	cufreeifnotnull(g->d.neff);
	cufreeifnotnull(g->d.q_spike);
	cufreeifnotnull(g->d.log_pq_spike);
	cufreeifnotnull(g->d.log_pq_nospike);
	cufreeifnotnull(g->d.nmean);
	cufreeifnotnull(g->d.gpmean);
	cufreeifnotnull(g->d.gpsqmean);
	cufreeifnotnull(g->d.O);
	cufreeifnotnull(g->d.Oi);
	cufreeifnotnull(g->d.ancestor);

	FreeCubTemporaryArrays(g); //calls cufreeifnotnull
}

int file2gpu(void * darray, FILE *pFile, int nbytes) {
	void * harray = malloc(nbytes);
	if (harray == NULL) //failed to allocate
			return 0;
	int nread = fread(harray, 1, nbytes, pFile);
	if (nread != nbytes) {
		free(harray);
		return -nread;
	}
	hipMemcpy(darray, harray, nbytes, hipMemcpyHostToDevice); //fixme: check for success here
	free(harray);
	return nread;
}

int save_gpu5sresults_toraw(FILE *pFile, gpu5s_problem * g) {
	//fixme save moments too
	float *ll = (float *) malloc(sizeof(float) * g->T);
	if (ll == NULL) {
		printf("Failed allocate memory!\n");
		return -2;
	}
	float *neff = (float *) malloc(sizeof(float) * g->T);
	if (neff == NULL) {
		printf("Failed allocate memory!\n");
		free(ll);
		return -3;
	}
	hipMemcpy(ll,   g->d.log_sum_raw_w, g->T * sizeof(float), hipMemcpyDeviceToHost); //fixme check for success
	hipMemcpy(neff, g->d.neff,          g->T * sizeof(float), hipMemcpyDeviceToHost); //fixme check for success
	if (fwrite(ll, sizeof(float),   g->T, pFile) != g->T) {
		printf("Failed to write sum of log weights!\n");
	}
	if (fwrite(neff, sizeof(float), g->T, pFile) != g->T) {
		printf("Failed to write neff!\n");
	}
	free(ll);
	free(neff);
	return 0;
}

gpu5s_problem * init_gpu5sproblem_fromraw(FILE *pFile) {
	int T, np, nsteps, K, ntimepoints_pre;
	if (fread((void *) &T, sizeof(int), 1, pFile) != 1) {
		return NULL;
	}
	if (fread((void *) &np, sizeof(int), 1, pFile) != 1) {
		return NULL;
	}
	if (fread((void *) &nsteps, sizeof(int), 1, pFile) != 1) {
		return NULL;
	}
	if (fread((void *) &K, sizeof(int), 1, pFile) != 1) {
		return NULL;
	}
	if (fread((void *) &ntimepoints_pre, sizeof(int), 1, pFile) != 1) {
		return NULL;
	}
	gpu5s_problem *g = (gpu5s_problem *) malloc(sizeof(gpu5s_problem));
	g->h.fobs = NULL;
	g->h.u = NULL;
	int maxKp1 = 1;
	while(maxKp1 < K + 1)
		maxKp1 *= 2; //set maxK so that maxK+1 is a power of 2 and maxK is sufficiently large
	int maxK = maxKp1 - 1;
	if (allocate_gpupfdata(g, T, np, nsteps, maxK, (K + 1) * nsteps, (T + ntimepoints_pre) * nsteps, ntimepoints_pre * nsteps) < 0) {
		std::cerr << "Allocation failed!\n";
		return NULL;
	}
	g->options.K = K;
	g->options.nsteps = nsteps;
	g->options.ntimepoints_pre = ntimepoints_pre;
	g->T = T;

	g->options.ntimepoints_pre = ntimepoints_pre;
	g->options.computenmean = true;
	if (fread((void *) &(g->dt), sizeof(float), 1, pFile) != 1) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (fread((void *) &(g->options.resamplethreshold), sizeof(float), 1, pFile) != 1) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (file2gpu(g->d.neff, pFile, sizeof(float)) != sizeof(float)) { //neff0
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (file2gpu(g->d.log_sum_raw_w, pFile, sizeof(float)) != sizeof(float)) { //log_sum_raw_w0
		destroy_gpu5s_problem(g);
		return NULL;
	}
	g->h.fobs   = (float *) malloc(sizeof(float) * g->T);
	g->h.u      = (float *) malloc(sizeof(float) * g->T);
	if (g->h.fobs == NULL || g->h.u == NULL) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (fread((void *) (g->h.fobs), sizeof(float), g->T, pFile) != g->T) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (fread((void *) (g->h.u), sizeof(float), g->T, pFile) != g->T) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	int nbytes_q = sizeof(float) * g->options.nsteps * (g->T + g->options.ntimepoints_pre);
	if (file2gpu(g->d.q_spike, pFile, nbytes_q) != nbytes_q) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (file2gpu(g->d.log_pq_spike, pFile, nbytes_q) != nbytes_q) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (file2gpu(g->d.log_pq_nospike, pFile, nbytes_q) != nbytes_q) {
		destroy_gpu5s_problem(g);
		return NULL;
	}

	if (fread((void *) &(g->params.db1), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.db2), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.db3), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.db4), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.vF), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.sigma_r), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.lambda), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.S), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.FBGp1), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.gain), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.kd_ex), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.maxex), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.c0), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.fdc), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.Btot0), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.Btot1), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.kon0), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.kon1), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.kon2), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.kon3), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.koff0), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.koff1), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.koff2), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.koff3), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.kon_B0), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.koff_B0), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.kon_B1), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }
	if (fread((void *) &(g->params.koff_B1), sizeof(float), 1, pFile) != 1) { destroy_gpu5s_problem(g); return NULL; }

	int state_nbytes = sizeof(float4) * g->options.nparticles;
	if (file2gpu(g->d.states, pFile, state_nbytes) != state_nbytes) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (file2gpu(g->d.cbr, pFile, state_nbytes) != state_nbytes) {
		destroy_gpu5s_problem(g);
		return NULL;
	}

	int npf_nbytes = sizeof(float) * g->options.nparticles;
	if (file2gpu(g->d.w, pFile, npf_nbytes) != npf_nbytes) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (file2gpu(g->d.log_w, pFile, npf_nbytes) != npf_nbytes) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (file2gpu(g->d.log_w_corrected, pFile, npf_nbytes) != npf_nbytes) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	int ns_pre_nbytes = g->options.nparticles * g->options.nsteps * g->options.ntimepoints_pre;
	if (file2gpu(g->d.ns_pre, pFile, ns_pre_nbytes) != ns_pre_nbytes) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	if (file2gpu(g->d.gp, pFile, npf_nbytes) != npf_nbytes) {
		destroy_gpu5s_problem(g);
		return NULL;
	}
	return g;
}
